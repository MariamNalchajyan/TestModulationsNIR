#include "hip/hip_runtime.h"

#include <stdio.h>
#include <hip/hip_runtime.h>

#include <vector>
#include "Generation.h"

#include <thrust/complex.h>


using std::generate;
using std::vector;



__global__ void Kernel(thrust::complex<double>* s1, double* keys1, thrust::complex<double>* s2, double* keys2,
    thrust::complex<double>* VFN, double* time_keys, double* freq_keys,
    int s1_size, int s2_size, int time_keys_size, int freq_keys_size, double sampling_frequency)
{
    int n = blockIdx.y * blockDim.y + threadIdx.y;
    int g = blockIdx.x * blockDim.x + threadIdx.x;
    double k = 1. + g * sampling_frequency / (s2_size - 1) / 1600.e6;//1600e6 - ��� f1

    thrust::complex<double> temp_sum(0, 0);
  
    if ((n < time_keys_size) && (g < freq_keys_size)) {

        for (int m = 0; m < s1_size; m++) {
            double index = (double)(m + n-s2_size) / k;
            int i1 = (int)index;
            int i2 = i1 + 1;
            if (i1 >= 0 && (i2 < s2_size))
            {
                double re = s2[i1].real() + (index / sampling_frequency - keys2[i1]) * (s2[i2].real() - s2[i1].real()) / (keys2[i2] - keys2[i1]);
                double im = s2[i1].imag() + (index / sampling_frequency - keys2[i1]) * (s2[i2].imag() - s2[i1].imag()) / (keys2[i2] - keys2[i1]);
                temp_sum += s1[m] * thrust::complex < double >(re, -im)* exp(thrust::complex<double>(0, -1) * 2. * M_PI * (k - 1) * 1600.e6 * (double)m / sampling_frequency);
            }
            else continue;

        }
        VFN[n * freq_keys_size + g] = temp_sum;
        freq_keys[g] = (g * sampling_frequency / (s2_size - 1));///*(k - 1) * 1600.e6*/);
        time_keys[n] = ((double)(n - s2_size) / sampling_frequency);
    }
}


void cudaModifVFN(Signal& signal1, Signal& signal2, Signal2D& outsignal, double sampling_frequency)
{
    outsignal.signal.clear();
    outsignal.t_keys.clear();
    outsignal.f_keys.clear();

    int N1 = signal1.keys.size();
    int N2 = signal2.keys.size();
    

    outsignal.signal.resize((N1 + N2));
    outsignal.t_keys.resize((N1 + N2));
    outsignal.f_keys.resize(N2);

    // Host vectors
    vector<thrust::complex<double>> h_s1(N1);
    vector<thrust::complex<double>> h_s2(N2);
    vector<double>h_k1 = signal1.keys;
    vector<double>h_k2 = signal2.keys;
    vector<thrust::complex<double>> h_vfn((N1 + N2) * N2);
    vector<double>h_time_keys((N1 + N2));
    vector<double>h_freq_keys(N2);

    for (int i = 0; i < N1; i++)
    {
        h_s1[i] = (thrust::complex<double>(signal1.signal[i].real(), signal1.signal[i].imag()));
    }
    for (int i = 0; i < N2; i++)
    {
        h_s2[i] = (thrust::complex<double>(signal2.signal[i].real(), signal2.signal[i].imag()));
    }

    // Allocate device memory
    thrust::complex<double>* d_s1, * d_s2, * d_vfn;
    double* d_k1, * d_k2;
    double* d_t, * d_f;
    hipMalloc(&d_s1, N1 * sizeof(thrust::complex<double>));
    hipMalloc(&d_s2, N2 * sizeof(thrust::complex<double>));
    hipMalloc(&d_k1, N1 * sizeof(double));
    hipMalloc(&d_k2, N2 * sizeof(double));
    hipMalloc(&d_vfn, (N1+N2) * N2 * sizeof(thrust::complex<double>));
    hipMalloc(&d_t, (N1 + N2) * sizeof(double));
    hipMalloc(&d_f, N2 * sizeof(double));

    // Copy data to the device
    hipMemcpy(d_s1, h_s1.data(), N1 * sizeof(thrust::complex<double>), hipMemcpyHostToDevice);
    hipMemcpy(d_s2, h_s2.data(), N2 * sizeof(thrust::complex<double>), hipMemcpyHostToDevice);
    hipMemcpy(d_k1, h_k1.data(), N1 *sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_k2, h_k2.data(), N2*sizeof(double), hipMemcpyHostToDevice);

    dim3 threads(32,32);
    dim3 blocks(N1/32, (N1 + N2)/32);

    // Launch kernel
    Kernel << <blocks, threads >> > (d_s1, d_k1, d_s2, d_k2, d_vfn, d_t, d_f, N1, N2, (N1 + N2), N2, sampling_frequency);

    // Copy back to the host
    hipMemcpy(h_vfn.data(), d_vfn, (N1 + N2) * N2 * sizeof(thrust::complex<double>), hipMemcpyDeviceToHost);
    hipMemcpy(h_time_keys.data(), d_t, (N1 + N2) * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_freq_keys.data(), d_f, N2 * sizeof(double), hipMemcpyDeviceToHost);

    for (int i = 0; i < (N1 + N2); i++)
    {
        outsignal.signal[i].resize(N2);
        for (int j = 0; j < N2; j++)
        {
            outsignal.signal[i][j] = h_vfn[i * N2 + j];
        }
    }
    outsignal.t_keys = h_time_keys;
    outsignal.f_keys = h_freq_keys;
    //int iter = 0;
    //for (int n = -(int)(signal2.signal.size()); n <= (int)(signal1.signal.size() - 1); n++)
    //{
    //    outsignal.t_keys[iter] = ((double)n / sampling_frequency);
    //    iter++;
    //}
    //for (int g = 0; g < signal2.signal.size(); g++)
    //{
    //    outsignal.f_keys[g] = (g * sampling_frequency / (signal2.signal.size() - 1)/*(k - 1) * 1600.e6*/);
    //}

    cutFrequency(outsignal);
    
    // Free memory on device
    hipFree(d_s1);
    hipFree(d_s2);
    hipFree(d_k1);
    hipFree(d_k2);
    hipFree(d_vfn);
    hipFree(d_t);
    hipFree(d_f);

 
}